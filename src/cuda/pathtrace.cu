#include "hip/hip_runtime.h"
#include "gpu_kernels.h"

#define RANDOM 12345

__device__
bool
intersect(ray_t& ray, shape_t& shape, hit_t* hit)
{
    float4 trans_origin = ray.pos - shape.sphere.origin;
    float a = dot(ray.dir, ray.dir);
    float b = 2 * dot(trans_origin, ray.dir);
    float c = dot(trans_origin, trans_origin) - shape.sphere.radius * shape.sphere.radius;

    float disc = b * b - 4 * a * c;
    if (disc < 0)       return false;

    // We use the following in place of the quadratic formula for
    // more numeric precision.
    float q = (b > 0) ?
            -0.5 * (b + sqrt(disc)) :
            -0.5 * (b - sqrt(disc));
    float t0 = q / a;
    float t1 = c / q;
    //if (t0 < t1) swap(t0,t1);

    float t;
    if (t0 < EPSILON)   return false;
    if (t1 < 0)         t = t0;
    else                t = t1;

    hit->pos    = ray.pos + t * ray.dir;
    hit->normal = normalize(hit->pos - shape.sphere.origin);
    hit->matidx = shape.matidx;

    return true;
}

__global__
void
pathtraceray(camera_t         cam,
             float4*          d_result,
             float4*          d_random,
             mat_t*           d_materials,
             ray_t*           d_raydirs,
             shape_t*         d_shapes, int num_shapes)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    int yi = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx = (yi * cam.width + xi);

    if (d_raydirs[idx].dir.w < 0) return;

    float dist = FLT_MAX;
    hit_t hit;
    for (int i = 0; i < num_shapes; i++)
    {
        hit_t new_hit;
        if (intersect(d_raydirs[idx], d_shapes[i], &new_hit))
        {
            float new_dist = length(new_hit.pos - cam.pos);
            if (new_dist < dist)
            {
                dist = new_dist;
                hit  = new_hit;
            }
        }
    }

    // Check whether this ray intersected the scene, if not kill the ray
    if (dist >= FLT_MAX)
    {
        d_result[idx]        = (float4){ 0, 0, 0, 0 };
        d_raydirs[idx].dir.w = -1;
        return;
    }
    mat_t* mat = d_materials + hit.matidx;

    // Check whether we hit a light, if so kill the ray
    if (mat->emit > 0)
    {
        // Should be: emission + color * (recursive path trace)
        d_result[idx]        = d_result[idx] + d_result[idx] * d_materials[hit.matidx].emit * d_materials[hit.matidx].color;
        d_raydirs[idx].dir.w = -1;
        return;
    }

    // Calculate color contribution
    d_result[idx] = d_result[idx] * d_materials[hit.matidx].color;

    // Reflect, refract or both
    if (mat->type == MIRROR)
    {
        float4 tempray = reflect(d_raydirs[idx].dir, hit.normal);
        d_raydirs[idx].dir = reflect(d_raydirs[idx].dir, hit.normal);
    }
    else if (mat->type == DIFFUSE)
    {
        unsigned randidx = (idx + (int)dot(d_raydirs[idx].dir, d_raydirs[idx].pos)) % (cam.width * cam.height);
        d_raydirs[idx].dir = randvector(d_random[randidx], hit.normal);
    }
    else if (mat->type == TRANSPARENT)
    {
        // calculate refraction and reflection
        printf("should not print!");
    }
    d_raydirs[idx].pos = hit.pos + EPSILON * d_raydirs[idx].dir;
}

int
cudapathtrace(camera_t        cam,
              float4*         d_result,
              float4*         d_random,
              mat_t*          d_materials,
              ray_t*          d_raydirs,
              shape_t*        d_shapes, int num_shapes)
{
    unsigned size = cam.height * cam.width;

    hiprandGenerator_t gen;
    SAFE_RAND( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
    SAFE_RAND( hiprandSetPseudoRandomGeneratorSeed(gen, RANDOM) );
    SAFE_RAND( hiprandGenerateUniform(gen, (float*)d_random, 4 * size) );

    // Perform computation on device
    dim3 threadsperblock(8, 8);
    dim3 numblocks(cam.width / threadsperblock.x,
                   cam.height / threadsperblock.y);
    for (int i = 0; i < 5; i++)
    {
        pathtraceray <<< numblocks,threadsperblock >>> (cam,
                                                        d_result,
                                                        d_random,
                                                        d_materials,
                                                        d_raydirs,
                                                        d_shapes, num_shapes);
    }
    CHECK_ERROR("Launching pathtrace kernel");

    return 0;
}

__global__
void
rgbtoint(camera_t      cam,
         float4*       d_result,
         color_t*      d_buffer)
{
    int xi = blockIdx.x * blockDim.x + threadIdx.x;
    int yi = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx = (yi * cam.width + xi);

    d_buffer[idx].r = d_result[idx].x * 255;
    d_buffer[idx].g = d_result[idx].y * 255;
    d_buffer[idx].b = d_result[idx].z * 255;
}

int cudargbtoint(camera_t        cam,
                 float4*         d_result,
                 color_t*        d_buffer)
{
    dim3 threadsperblock(8, 8);
    dim3 numblocks(cam.width / threadsperblock.x,
                   cam.height / threadsperblock.y);
    rgbtoint <<< numblocks,threadsperblock >>> (cam,
                                                d_result,
                                                d_buffer);
    CHECK_ERROR("Launching rgbtoint kernel");

    return 0;
}
