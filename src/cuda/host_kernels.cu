#include "device_util.cuh"
#include "host_kernels.cuh"

__host__
int
cudamallocshapes(shape_t*& d_shapes,
                 shape_t* shapes,
                 int size)
{
    SAFE(hipMalloc(&d_shapes, sizeof(shape_t)));
    SAFE(hipMemcpy(d_shapes, shapes, size * sizeof(shape_t), hipMemcpyHostToDevice));

    return 0;
}

__host__
int
cudamallocbuffer(unsigned char*& d_buffer,
                 int size)
{
    SAFE(hipMalloc(&d_buffer, 3 * size * sizeof(unsigned char)));

    return 0;
}

__host__
int
cudareadbuffer(unsigned char* buffer,
               unsigned char* d_buffer,
               int size)
{
    // Read results
    SAFE(hipMemcpy(buffer, d_buffer, 3 * size * sizeof(unsigned char), hipMemcpyDeviceToHost));

    return 0;
}
